#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include "NvAnalysis.h"

#define BOX_W 32
#define BOX_H 32

__global__ void
addLabelsKernel(int *pDevPtr, int pitch)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y + BOX_H;
    int col = blockIdx.x * blockDim.x + threadIdx.x + BOX_W;
    char *pElement = (char *)pDevPtr + row * pitch + col;

    pElement[0] = 0;

    return;
}

int
addLabels(hipDeviceptr_t pDevPtr, int pitch)
{
    dim3 threadsPerBlock(BOX_W, BOX_H);
    dim3 blocks(1,1);

    addLabelsKernel<<<blocks,threadsPerBlock>>>((int *)pDevPtr, pitch);

    return 0;
}


__global__ void
convertIntToFloatKernel(hipDeviceptr_t pDevPtr, int width, int height,
                void* cuda_buf, int pitch)
{
    float *pdata = (float *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width && row < height)
    {
        for (int k = 0; k < 3; k++)
        {
            pdata[width * height * k + row * width + col] =
                (float)*(psrcdata + row * pitch + col * 4 + (3 - 1 - k));
        }
    }
}

int convertIntToFloat(hipDeviceptr_t pDevPtr, int width, int height,
        void* cuda_buf, int pitch)
{
    dim3 threadsPerBlock(32, 32);
    dim3 blocks(width/threadsPerBlock.x, height/threadsPerBlock.y);

    convertIntToFloatKernel<<<blocks, threadsPerBlock>>>(pDevPtr, width,
                height, cuda_buf, pitch);

    return 0;
}