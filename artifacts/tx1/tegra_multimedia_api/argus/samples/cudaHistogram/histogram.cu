#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Based on CUB histogram code: https://github.com/NVlabs/cub/tree/master/experimental/histogram
 */

#include "histogram.h"

// First-pass histogram kernel (binning into privatized counters)
template <
    int         NUM_PARTS,
    int         NUM_BINS>
__global__ void histogram_smem_atomics(
    hipSurfaceObject_t surface,
    unsigned int width,
    unsigned int height,
    unsigned int *out)
{
    // global position and size
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    // threads in workgroup
    int t = threadIdx.x + threadIdx.y * blockDim.x; // thread index in workgroup, linear in 0..nt-1
    int nt = blockDim.x * blockDim.y; // total threads in workgroup

    // group index in 0..ngroups-1
    int g = blockIdx.x + blockIdx.y * gridDim.x;

    // initialize smem
    __shared__ unsigned int smem[NUM_BINS];
    for (int i = t; i < NUM_BINS; i += nt)
        smem[i] = 0;

    // process pixels (updates our group's partial histogram in smem)
    for (int col = x; col < width; col += nx)
    {
        for (int row = y; row < height; row += ny)
        {
            uchar1 data;
            surf2Dread(&data, surface, col, row);

            atomicAdd(&smem[((unsigned int)data.x) % NUM_BINS], 1);
        }
    }

    __syncthreads();

    // move to our workgroup's slice of output
    out += g * NUM_PARTS;

    // store local output to global
    for (int i = t; i < NUM_BINS; i += nt)
    {
        out[i] = smem[i];
    }
}

// Second pass histogram kernel (accumulation)
template <
    int         NUM_PARTS,
    int         NUM_BINS>
__global__ void histogram_smem_accum(
    const unsigned int *in,
    int n,
    unsigned int *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > NUM_BINS)
        return; // out of range

    unsigned int total = 0;
    for (int j = 0; j < n; j++)
        total += in[i + NUM_PARTS * j];

    out[i] = total;
}

template <
    int         NUM_BINS>
float run_smem_atomics(
    hipSurfaceObject_t surface,
    unsigned int width,
    unsigned int height,
    unsigned int *h_hist)
{
    enum
    {
        NUM_PARTS = 1024
    };

    dim3 block(32, 4);
    dim3 grid(16, 16);
    int total_blocks = grid.x * grid.y;

    // allocate device histogram
    unsigned int *d_hist;
    hipMalloc(&d_hist, NUM_BINS * sizeof(unsigned int));
    // allocate partial histogram
    unsigned int *d_part_hist;
    hipMalloc(&d_part_hist, total_blocks * NUM_PARTS * sizeof(unsigned int));

    dim3 block2(128);
    dim3 grid2((NUM_BINS + block.x - 1) / block.x);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&stop);
    hipEventCreate(&start);

    hipEventRecord(start, 0);

    histogram_smem_atomics<NUM_PARTS, NUM_BINS><<<grid, block>>>(
        surface,
        width,
        height,
        d_part_hist);

    histogram_smem_accum<NUM_PARTS, NUM_BINS><<<grid2, block2>>>(
        d_part_hist,
        total_blocks,
        d_hist);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float elapsed_millis;
    hipEventElapsedTime(&elapsed_millis, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_hist, d_hist, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(d_part_hist);
    hipFree(d_hist);

    return elapsed_millis;
}

float histogram(hipSurfaceObject_t surface, unsigned int width, unsigned int height,
    unsigned int *histogram)
{
    return run_smem_atomics<HISTOGRAM_BINS>(surface, width, height, histogram);
}
