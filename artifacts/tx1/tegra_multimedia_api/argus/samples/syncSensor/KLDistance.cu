#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _KLDISTANCE_KERNEL_H_
#define _KLDISTANCE_KERNEL_H_

#include <stdio.h>
#include "KLDistance.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the KL ratio from probability ratios.
 *
 */
__global__ void
vectorKLRatio(const unsigned int *A,
              const unsigned int *B,
              float *C,
              const int numElements,
              const int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        float a = A[i];
        float b = B[i];
        a = a/(float)size;
        b = b/(float)size;
        if ( b == 0.0f) b+= .0001; // add sigma
        if ( a != 0)
            C[i] = a * log(a/b);
        else
            C[i] = 0.0f;
    }
}

float computeKLDistance(unsigned int *histOne,
                        unsigned int *histTwo,
                        const unsigned int bins,
                        const unsigned int size,
                        float *distance)
{
    hipError_t err = hipSuccess;
    int threadsPerBlock = 256;
    int blocksPerGrid =(bins + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&stop);
    hipEventCreate(&start);

    hipEventRecord(start, 0);
    unsigned int *d_histOne = NULL;
    err = hipMalloc((void **)&d_histOne, bins * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device histOne (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    unsigned int *d_histTwo = NULL;
    err = hipMalloc((void **)&d_histTwo, bins * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device histTwo (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the ratio bins
    float *ratio = NULL;
    err = hipMalloc((void **)&ratio, bins * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector ratio (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_histOne, histOne, bins, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy into device vector histOne (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_histTwo, histTwo, bins, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy into device vector histTwo (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    vectorKLRatio<<<blocksPerGrid, threadsPerBlock>>>(d_histOne, d_histTwo, ratio, bins, size);
    if (hipGetLastError() != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *dispRatio = (float *)malloc(bins * sizeof(float));
    hipMemcpy(dispRatio, ratio, bins * sizeof(float), hipMemcpyDeviceToHost);

    // just sum up the vector.
    *distance = 0.0f;
    for (unsigned int i = 0; i < bins; ++i)
    {
        *distance += dispRatio[i];
    }

    err = hipFree(d_histOne);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector histOne (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_histTwo);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector histTwo (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(ratio);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector ratio (error code %s)!\n",
                        hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(dispRatio);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed_millis;
    hipEventElapsedTime(&elapsed_millis, start, stop);
    return elapsed_millis;
}
#endif
