#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <cstdio>
#include "NvAnalysis.h"

#define BOX_W 32
#define BOX_H 32

__global__ void
addLabelsKernel(int *pDevPtr, int pitch)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y + BOX_H;
    int col = blockIdx.x * blockDim.x + threadIdx.x + BOX_W;
    char *pElement = (char *)pDevPtr + row * pitch + col;

    pElement[0] = 0;

    return;
}

int
addLabels(hipDeviceptr_t pDevPtr, int pitch)
{
    dim3 threadsPerBlock(BOX_W, BOX_H);
    dim3 blocks(1,1);

    addLabelsKernel<<<blocks,threadsPerBlock>>>((int *)pDevPtr, pitch);

    return 0;
}


__global__ void
convertIntToFloatKernel(hipDeviceptr_t pDevPtr, int width, int height,
                void* cuda_buf, int pitch)
{
    float *pdata = (float *)cuda_buf;
    char *psrcdata = (char *)pDevPtr;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width && row < height)
    {
        // Distinct planes of B, G and R components
        for (int k = 0; k < 3; k++)
        {
            pdata[width * height * k + row * width + col] =
                (float)*(psrcdata + row * pitch + col * 4 + (3 - 1 - k));
        }
    }
}

__global__ void
convertIntToFloatWithMeanKernel(
    int* rgba,
    int width,
    int height,
    void* cuda_buf,
    int pitch,
    float3 mean)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = width * height;
	
	if( x >= width || y >= height )
		return;

    char* input = (char*)rgba;
    float* output = (float*) cuda_buf;
    int pxStart = (y * width + x) * 4;

#if 0
    // first one is alpha
	output[n * 0 + y * width + x] = input[pxStart + 1] - mean.x; //< Blue
	output[n * 1 + y * width + x] = input[pxStart + 2] - mean.y; //< Green
	output[n * 2 + y * width + x] = input[pxStart + 3] - mean.z; //< Red
#else
	output[n * 0 + y * width + x] = input[pxStart + 2] - mean.x; //< Blue
	output[n * 1 + y * width + x] = input[pxStart + 1] - mean.y; //< Green
	output[n * 2 + y * width + x] = input[pxStart + 0] - mean.z; //< Red
#endif

#if 0
    if ((x == 0 && y == 0))
    {
        for (int i =0 ; i < 100; i++)
        {
            printf("KERNEL INPUT %i %i\n", i, input[i]);
        }   
    }
#endif
}


int convertIntToFloat(hipDeviceptr_t pDevPtr, int width, int height,
        void* cuda_buf, int pitch)
{
    dim3 threadsPerBlock(32, 32);
    dim3 blocks(width/threadsPerBlock.x, height/threadsPerBlock.y);

    convertIntToFloatKernel<<<blocks, threadsPerBlock>>>(
        pDevPtr,
        width,
        height,
        cuda_buf,
        pitch);

    return 0;
}

int convertIntToFloatWithMean(
    int* rgbaDevicePtr,
    int width,
    int height,
    void* cuda_buf,
    int pitch,
    float3 mean)
{
    dim3 threadsPerBlock(32, 32);
    dim3 blocks(width/threadsPerBlock.x, height/threadsPerBlock.y);

#if 0
    printf("Converting int to float with mean: width %i, height %i, pitch %i\n",
        width,
        height,
        pitch);
#endif

    convertIntToFloatWithMeanKernel<<<blocks, threadsPerBlock>>>(
        rgbaDevicePtr,
        width,
        height,
        cuda_buf,
        pitch,
        mean);

    return 0;
}

